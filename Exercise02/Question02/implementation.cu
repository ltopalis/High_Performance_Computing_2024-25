#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define N 10

int *malloc_h(long unsigned int dim);
bool check_arrays(int *a, int *b, long unsigned int dim);
void matrix_mul_cpu(int *a, int *b, int *c, int *d, int *e, int *f, int n);

__global__ void matrix_mul(int *a, int *b, int *c, int *d, int *e, int *f, int n);

int main()
{
    int *a, *b, *c, *d, *e, *f, *e_cpu, *f_cpu; // host
    int *d_a, *d_b, *d_c, *d_d, *d_e, *d_f;     // device

    // memory allocation
    a = malloc_h(N);
    b = malloc_h(N);
    c = malloc_h(N);
    d = malloc_h(N);
    e = malloc_h(N);
    f = malloc_h(N);
    e_cpu = malloc_h(N);
    f_cpu = malloc_h(N);

    hipMalloc((void **)&d_a, N * N * sizeof(int));
    hipMalloc((void **)&d_b, N * N * sizeof(int));
    hipMalloc((void **)&d_c, N * N * sizeof(int));
    hipMalloc((void **)&d_d, N * N * sizeof(int));
    hipMalloc((void **)&d_e, N * N * sizeof(int));
    hipMalloc((void **)&d_f, N * N * sizeof(int));

    // initialize matrices
    srand(time(NULL));
    for (int i = 0; i < N * N; i++)
    {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
        c[i] = rand() % 10;
        d[i] = rand() % 10;
        e[i] = 0;
        f[i] = 0;
        e_cpu[i] = 0;
        f_cpu[i] = 0;
    }

    hipMemcpy(d_a, a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_d, d, N * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    matrix_mul<<<gridSize, blockSize>>>(d_a, d_b, d_c, d_d, d_e, d_f, N);
    hipDeviceSynchronize();

    hipMemcpy(e, d_e, N * N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(f, d_f, N * N * sizeof(int), hipMemcpyDeviceToHost);

    matrix_mul_cpu(a, b, c, d, e_cpu, f_cpu, N);

    bool result = check_arrays(e, e_cpu, N) & check_arrays(f, f_cpu, N);
    if (result)
        printf("Results match!\n");
    else
        printf("Results mismatch!\n");

    // Free memory
    free(a);
    free(b);
    free(c);
    free(d);
    free(e);
    free(f);
    free(e_cpu);
    free(f_cpu);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);
    hipFree(d_e);
    hipFree(d_f);

    return 0;
}

int *malloc_h(long unsigned int dim)
{
    int *p = (int *)malloc(dim * dim * sizeof(int));
    if (!p)
    {
        fprintf(stderr, "Host memory allocation failed.\n");
        exit(0);
    }
    return p;
}

bool check_arrays(int *a, int *b, long unsigned int dim)
{
    for (long unsigned int i = 0; i < dim * dim; i++)
    {
        if (a[i] != b[i])
        {
            printf("Mismatch at index %lu: GPU = %d, CPU = %d\n", i, a[i], b[i]);
            return false;
        }
    }
    return true;
}

void matrix_mul_cpu(int *a, int *b, int *c, int *d, int *e, int *f, int n)
{

    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            int sum_r = 0;
            int sum_i = 0;

            for (int k = 0; k < n; k++)
            {
                sum_r += a[i * n + k] * b[k * n + j];
                sum_i += c[i * n + k] * d[k * n + j];
            }
            e[i * n + j] = sum_r;
            f[i * n + j] = sum_i;
        }
    }
}

__global__ void matrix_mul(int *a, int *b, int *c, int *d, int *e, int *f, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n)
    {
        int sum_r = 0;
        int sum_i = 0;
        for (int i = 0; i < n; i++)
        {
            sum_r += a[row * n + i] * b[i * n + col];
            sum_i += c[row * n + i] * d[i * n + col];
        }
        e[row * n + col] = sum_r;
        f[row * n + col] = sum_i;
    }
}
